#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <math.h>
#include <string.h>
#include <iostream>
#include <fstream>


#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "GPUProfiler.h"
#include "Printers.h"

#include "Iterations.h"


///
/// \brief CUFFT_2D_C2C
/// \param NX
/// \param NY
/// \param data
/// \param iteration
///
void CUFFT_2D_C2C
(const unsigned int NX, const unsigned int NY,
 hipfftComplex* data, const unsigned int iteration)
{
    // CUFFT plan
    hipfftHandle plan;

    // Create a 2D FFT plan
    hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C);

    char profileName [2048] = "b_CUFFT_2D_C2C";
    char N_char[1024];
    sprintf(N_char, "__%dx%d__%d", NX, NY, iteration);
    strcat(profileName, N_char);

    // Plan execution, transform the first signal in place
    START_GPU_PROFILING(profileName);
    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
    END_GPU_PROFILING();

    // Destroy the cuFFT plan
    hipfftDestroy(plan);
}

///
/// \brief main
/// \param argc
/// \param argv
/// \return
///
int main(int argc, char** argv)
{
    for (unsigned int N = INITIAL_SIZE_2D; N <= LIMIT_2D; INCREMENT)
    {
        for (unsigned int itr = 0; itr < NUM_ITERATIONS; itr++)
        {
            const unsigned int NX = N;
            const unsigned int NY = N;

            // Data size
            const size_t dataSizeCUFFT = sizeof(hipfftComplex)* NX * NY;

            // CPU allocation
            hipfftComplex *dataCPU;
            dataCPU = (hipfftComplex *) malloc(dataSizeCUFFT);

            // GPU allocation
            hipfftComplex* dataGPU;
            hipMalloc((void**)&dataGPU, dataSizeCUFFT);

            // Initialize CPU array with random numbers
            for (unsigned int i = 0; i < NX * NY; i++)
            {
                dataCPU[i].x = float(i % 256) + i * 0.00001;
                dataCPU[i].y = float(i % 256) + i * 0.00002;
            }

            // Upload the random array to the GPU
            hipMemcpy(dataGPU, dataCPU, dataSizeCUFFT, hipMemcpyHostToDevice);

            // Ensure empty array ready to recieve the results
            for (unsigned int i = 0; i < NX * NY; i++)
            {
                dataCPU[i].x = 0.f;
                dataCPU[i].y = 0.f;
            }

            // Execute the kernel
            CUFFT_2D_C2C(NX, NY, dataGPU, itr);

            // Download the results to the CPU array
            hipMemcpy(dataCPU, dataGPU, dataSizeCUFFT, hipMemcpyDeviceToHost);

            // Release the data on the GPU
            hipFree(dataGPU);

            // Write the data to a file for a single iteration only
            if (NUM_ITERATIONS == 1)
            {
                std::ofstream fileStream;
                char fileName[1024];
                sprintf(fileName, "c_CUFFT_2D_C2C__%dx%d.check", NX, NY);
                fileStream.open(fileName);

                unsigned int index = 0;
                for (unsigned int i = 0; i < NX; i++)
                    for (unsigned int j = 0; j < NY; j++)
                    {
                        if (index < 8 || index > NX * NY - 8)
                            fileStream << i << "," << j <<":"
                                       << dataCPU[index].x << ","
                                       << dataCPU[index].y << std::endl;
                        index++;
                    }

                fileStream.close();
            }

            // Release the data on the CPU
            free(dataCPU);
        }
    }

    return 0;
}
